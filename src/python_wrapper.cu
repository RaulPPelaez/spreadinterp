#include "hip/hip_runtime.h"
#include "utils.cuh"
#include <nanobind/nanobind.h>
#include <nanobind/ndarray.h>
#include <nanobind/stl/optional.h>
#include <optional>
#include <thrust/copy.h>
#include <thrust/device_vector.h>

namespace nb = nanobind;
using namespace nb::literals;
using uammd::real;

using pyarray_c =
    nb::ndarray<real, nb::c_contig, nb::shape<-1, -1>, nb::device::cuda>;
using pyarray3_c =
    nb::ndarray<real, nb::c_contig, nb::shape<-1, 3>, nb::device::cuda>;
using pyarray_field_c =
    nb::ndarray<real, nb::c_contig, nb::shape<-1, -1, -1, -1>,
                nb::device::cuda>;
using pyarray3f = nb::ndarray<real, nb::shape<3>, nb::device::cpu>;
using pyarray3i = nb::ndarray<int, nb::shape<3>, nb::device::cpu>;

using KernelBase = IBM_kernels::Peskin::threePoint;
struct Kernel {
  static constexpr int support = 3;
  Kernel(real3 h, bool is2D = false)
      : m_phiX(h.x), m_phiY(h.y), m_phiZ(h.z), is2D(is2D) {}

  __host__ __device__ real phiX(real rr, real3 pos = real3()) const {
    return m_phiX.phi(rr, pos);
  }

  __host__ __device__ real phiY(real rr, real3 pos = real3()) const {
    return m_phiY.phi(rr, pos);
  }

  __host__ __device__ real phiZ(real rr, real3 pos = real3()) const {
    return is2D ? real(1.0) : m_phiZ.phi(rr, pos);
  }

private:
  KernelBase m_phiX, m_phiY, m_phiZ;
  bool is2D;
};

struct threePointDerivative {
  const real invh;
  static constexpr int support = 3;
  threePointDerivative(real h) : invh(1.0 / h) {}

  __host__ __device__ real phi(real rr, real3 pos = real3()) const {
    real r = fabs(rr) * invh;
    real sgn = (rr >= 0) ? 1.0 : -1.0;

    if (r < real(0.5)) {
      return -invh * invh * r * sgn / sqrt(real(1.0) - real(3.0) * r * r);
    } else if (r < real(1.5)) {
      return -invh * invh * (real(1.0) / real(2.0)) *
             (real(1.0) +
              (real(1.0) - r) / sqrt(real(1.0) - real(3.0) * (real(1.0) - r) *
                                                     (real(1.0) - r))) *
             sgn;
    } else {
      return 0;
    }
  }
};

struct GradientKernel {
  static constexpr int support = 3;

  GradientKernel(real3 h, bool is2D)
      : m_phiX(h.x), m_phiY(h.y), m_phiZ(h.z), m_dphiX(h.x), m_dphiY(h.y),
        m_dphiZ(h.z), is2D(is2D) {}

  __host__ __device__ std::tuple<real, real> phiX(real r,
                                                  real3 pos = real3()) const {
    return {m_phiX.phi(r, pos), m_dphiX.phi(r, pos)};
  }

  __host__ __device__ std::tuple<real, real> phiY(real r,
                                                  real3 pos = real3()) const {
    return {m_phiY.phi(r, pos), m_dphiY.phi(r, pos)};
  }

  __host__ __device__ std::tuple<real, real> phiZ(real r,
                                                  real3 pos = real3()) const {
    return {is2D ? real(1.0) : m_phiZ.phi(r, pos),
            is2D ? real(0.0) : m_dphiZ.phi(r, pos)};
  }

private:
  KernelBase m_phiX, m_phiY, m_phiZ;
  threePointDerivative m_dphiX, m_dphiY, m_dphiZ;
  bool is2D;
};

struct GradientInterpolationWeightCompute {
  template <typename T2>
  inline __device__ real3 operator()(real quantity,
                                     thrust::tuple<T2, T2, T2> kernel) const {
    auto [phiX, dphiX] = thrust::get<0>(kernel);
    auto [phiY, dphiY] = thrust::get<1>(kernel);
    auto [phiZ, dphiZ] = thrust::get<2>(kernel);
    real3 delta = {phiY * phiZ * dphiX, phiX * phiZ * dphiY,
                   phiX * phiY * dphiZ};
    return delta * quantity;
  }
};

struct GradientSpreadWeightCompute {
  template <class T2>
  inline __device__ real3 operator()(thrust::tuple<real3, real3> iquantity,
                                     thrust::tuple<T2, T2, T2> kernel) const {
    auto [phiX, dphiX] = thrust::get<0>(kernel);
    auto [phiY, dphiY] = thrust::get<1>(kernel);
    auto [phiZ, dphiZ] = thrust::get<2>(kernel);
    real3 quantity = thrust::get<1>(iquantity);
    real3 direction = thrust::get<0>(iquantity);
    real delta = phiY * phiZ * dphiX * direction.x +
                 phiX * phiZ * dphiY * direction.y +
                 phiX * phiY * dphiZ * direction.z;
    return delta * quantity;
  }
};

struct LinearIndex3D {
  LinearIndex3D(int nx, int ny, int nz) : nx(nx), ny(ny), nz(nz) {}

  inline __device__ __host__ int operator()(int3 c) const {
    return this->operator()(c.x, c.y, c.z);
  }

  inline __device__ __host__ int operator()(int i, int j, int k) const {
    return k + nz * (j + ny * i);
  }

private:
  const int nx, ny, nz;
};

struct Permute {
  Permute(int nf, int i) : nf(nf), i(i) {}

  inline __device__ __host__ int operator()(int j) const { return j * nf + i; }

private:
  int nf, i;
};

void cudaCheckError() {
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    std::cerr << "CUDA error: " << hipGetErrorString(err) << std::endl;
    throw std::runtime_error("CUDA error: " +
                             std::string(hipGetErrorString(err)));
  }
}

void dispatchWithReal(auto &foo, pyarray_field_c ifield, pyarray_c iquantity) {
  if (iquantity.shape(1) == 1) {
    foo(reinterpret_cast<real *>(iquantity.data()),
        reinterpret_cast<real *>(ifield.data()));
  } else if (iquantity.shape(1) == 2) {
    foo(reinterpret_cast<real2 *>(iquantity.data()),
        reinterpret_cast<real2 *>(ifield.data()));
  } else if (iquantity.shape(1) == 3) {
    foo(reinterpret_cast<real3 *>(iquantity.data()),
        reinterpret_cast<real3 *>(ifield.data()));
  } else if (iquantity.shape(1) == 4) {
    foo(reinterpret_cast<real4 *>(iquantity.data()),
        reinterpret_cast<real4 *>(ifield.data()));
  } else {
    auto f_ptr = reinterpret_cast<real *>(ifield.data());
    auto q_ptr = reinterpret_cast<real *>(iquantity.data());
    int nf = ifield.shape(3);
    for (int i = 0; i < iquantity.shape(1); i++) {
      auto f_it = thrust::make_permutation_iterator(
          f_ptr, thrust::make_transform_iterator(
                     thrust::make_counting_iterator(0), Permute(nf, i)));
      auto q_it = thrust::make_permutation_iterator(
          q_ptr, thrust::make_transform_iterator(
                     thrust::make_counting_iterator(0), Permute(nf, i)));
      foo(q_it, f_it);
    }
  }
}

void interpolateField_direct(pyarray3_c ipos, pyarray_field_c ifield,
                             pyarray_c iquantity, real3 L) {
  const auto ni = ifield.shape_ptr();
  const int3 n = {int(ni[0]), int(ni[1]), int(ni[2])};
  const real3 cellSize = L / make_real3(n);
  auto kernel = std::make_shared<Kernel>(cellSize, n.z == 1);
  Grid grid(Box(L), n);
  IBM<Kernel, Grid, LinearIndex3D> ibm(kernel, grid);
  auto gather = [&](auto q_it, auto f_it) {
    ibm.gather(reinterpret_cast<real3 *>(ipos.data()), q_it, f_it,
               int(ipos.shape(0)));
  };
  dispatchWithReal(gather, ifield, iquantity);
  cudaCheckError();
}

struct Dot {
  inline __device__ __host__ real operator()(real3 a, real3 b) const {
    return a.x * b.x + a.y * b.y + a.z * b.z;
  }
};

void interpolateField_gradient(pyarray3_c ipos, pyarray_field_c ifield,
                               pyarray_c iquantity, pyarray3_c idirection,
                               real3 L) {
  if (iquantity.shape(1) != 3) {
    throw std::runtime_error("Quantity must be 3D");
  }
  if (idirection.shape(0) != ipos.shape(0)) {
    throw std::runtime_error(
        "Gradient direction must have same number of particles as pos");
  }
  const auto ni = ifield.shape_ptr();
  const int3 n = {int(ni[0]), int(ni[1]), int(ni[2])};
  const real3 cellSize = L / make_real3(n);
  auto kernel = std::make_shared<GradientKernel>(cellSize, n.z == 1);
  Grid grid(Box(L), n);
  IBM<GradientKernel, Grid, LinearIndex3D> ibm(kernel, grid);
  auto wc = GradientInterpolationWeightCompute();
  auto qw = IBM_ns::DefaultQuadratureWeights();
  auto q_it = reinterpret_cast<real *>(iquantity.data());
  auto d_ptr = reinterpret_cast<real3 *>(idirection.data());
  thrust::device_vector<real3> qd(ipos.shape(0));
  for (int i = 0; i < ifield.shape(3); i++) {
    thrust::fill(thrust::cuda::par, qd.begin(), qd.end(), real3());
    auto f_a = thrust::make_permutation_iterator(
        reinterpret_cast<real *>(ifield.data()),
        thrust::make_transform_iterator(thrust::make_counting_iterator(0),
                                        Permute(ifield.shape(3), i)));
    ibm.gather(reinterpret_cast<real3 *>(ipos.data()), qd.data().get(), f_a, qw,
               wc, int(ipos.shape(0)));
    auto qa_it = thrust::make_permutation_iterator(
        q_it, thrust::make_transform_iterator(thrust::make_counting_iterator(0),
                                              Permute(3, i)));
    thrust::transform(thrust::cuda::par, qd.begin(), qd.end(), d_ptr, qa_it,
                      Dot());
  }

  cudaCheckError();
}
void interpolateField_wrapper(pyarray3_c ipos, pyarray_field_c ifield,
                              pyarray_c iquantity, pyarray3f Li, bool gradient,
                              std::optional<pyarray3_c> gradient_direction) {
  if (ipos.shape(0) != iquantity.shape(0)) {
    throw std::runtime_error("Quantity shape does not match pos");
  }
  if (iquantity.shape(1) != ifield.shape(3)) {
    throw std::runtime_error("Quantity shape does not match field");
  }
  real3 L = {Li.view()(0), Li.view()(1), Li.view()(2)};
  if (!gradient) {
    interpolateField_direct(ipos, ifield, iquantity, L);
  } else {
    if (!gradient_direction.has_value()) {
      throw std::runtime_error("Gradient direction must be provided");
    }
    interpolateField_gradient(ipos, ifield, iquantity,
                              gradient_direction.value(), L);
  }
}

void spreadParticles_direct(pyarray3_c ipos, pyarray_c iquantity,
                            pyarray_field_c ifield, real3 L, int3 n) {
  real3 cellSize = L / make_real3(n);
  auto kernel = std::make_shared<Kernel>(cellSize, n.z == 1);
  Grid grid(Box(L), n);
  IBM<Kernel, Grid, LinearIndex3D> ibm(kernel, grid);
  auto spread = [&](auto q_it, auto f_it) {
    ibm.spread(reinterpret_cast<real3 *>(ipos.data()), q_it, f_it,
               int(ipos.shape(0)));
  };
  dispatchWithReal(spread, ifield, iquantity);
  cudaCheckError();
}

void spreadParticles_gradient(pyarray3_c ipos, pyarray_c iquantity,
                              pyarray_field_c ifield, pyarray3_c idirection,
                              real3 L, int3 n) {
  if (iquantity.shape(1) != 3) {
    throw std::runtime_error("Quantity must be 3D");
  }
  if (idirection.shape(0) != ipos.shape(0)) {
    throw std::runtime_error(
        "Gradient direction must have same number of particles as pos");
  }
  real3 cellSize = L / make_real3(n);
  auto kernel = std::make_shared<GradientKernel>(cellSize, n.z == 1);
  Grid grid(Box(L), n);
  IBM<GradientKernel, Grid, LinearIndex3D> ibm(kernel, grid);
  auto wc = GradientSpreadWeightCompute();
  auto q_it = reinterpret_cast<real3 *>(iquantity.data());
  auto d_ptr = reinterpret_cast<real3 *>(idirection.data());
  auto dq_it = thrust::make_zip_iterator(thrust::make_tuple(d_ptr, q_it));
  auto f_it = reinterpret_cast<real3 *>(ifield.data());
  ibm.spread(reinterpret_cast<real3 *>(ipos.data()), dq_it, f_it, wc,
             int(ipos.shape(0)));
  cudaCheckError();
}

void spreadParticles_wrapper(pyarray3_c ipos, pyarray_c iquantity,
                             pyarray_field_c ifield, pyarray3f Li, pyarray3i ni,
                             bool gradient,
                             std::optional<pyarray3_c> gradient_direction) {
  if (ipos.shape(0) != iquantity.shape(0)) {
    throw std::runtime_error("Quantity shape does not match pos");
  }
  if (iquantity.shape(1) != ifield.shape(3)) {
    throw std::runtime_error("Quantity shape does not match field");
  }
  real3 L = {Li.view()(0), Li.view()(1), Li.view()(2)};
  int3 n = {ni.view()(0), ni.view()(1), ni.view()(2)};
  if (!gradient)
    spreadParticles_direct(ipos, iquantity, ifield, L, n);
  else
    spreadParticles_gradient(ipos, iquantity, ifield,
                             gradient_direction.value(), L, n);
}

NB_MODULE(_spreadinterp, m) {
  m.def("interpolateField", &interpolateField_wrapper, "pos"_a, "field"_a,
        "quantity"_a.noconvert(), "L"_a, "gradient"_a = false,
        "gradient_direction"_a = nb::none());
  m.def("spreadParticles", &spreadParticles_wrapper, "pos"_a, "quantity"_a,
        "field"_a.noconvert(), "L"_a, "n"_a, "gradient"_a = false,
        "gradient_direction"_a = nb::none());
}
